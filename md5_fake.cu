
#include <hip/hip_runtime.h>
#include <stdint.h>

__device__ void fake_md5Hash(uint32_t* answer, uint8_t* data, uint32_t length, uint32_t *a1, uint32_t *b1, uint32_t *c1, uint32_t *d1) {
  uint8_t target[] = {' ', ' ', ' ', ' ', ' '};
  ssize_t len = sizeof(target);

  if (len != length) {
    *a1 = *b1 = *c1 = *d1 = 0;
    return;
  }

  for (auto i = 0; i < len; ++i) {
    if (data[i] != target[i]) {
      *a1 = *b1 = *c1 = *d1 = 0;
      return;
    }
  }

  *a1 = answer[0];
  *b1 = answer[1];
  *c1 = answer[2];
  *d1 = answer[3];
}
