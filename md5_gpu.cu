#include "hip/hip_runtime.h"
/**
 * CUDA MD5 cracker
 * Copyright (C) 2015  Konrad Kusnierz <iryont@gmail.com>
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <stdio.h>
#include <iostream>
#include <time.h>
#include <string.h>
#include <stdlib.h>
#include <stdint.h>
#include <sstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#define CONST_WORD_LIMIT 10
#define CONST_CHARSET_LIMIT 100

#define CONST_CHARSET_LENGTH 256

#define CONST_WORD_LENGTH_MIN 1
#define CONST_WORD_LENGTH_MAX 8

#define TOTAL_BLOCKS 16384UL
#define TOTAL_THREADS 512UL
#define HASHES_PER_KERNEL 256UL

#include "assert.cu"
#include "md5.cu"
#include "md5_fake.cu"

/* Global variables */
uint8_t g_wordLength;

uint8_t g_word[CONST_WORD_LIMIT];
uint8_t g_cracked[CONST_WORD_LIMIT];

__device__ uint8_t g_deviceCracked[CONST_WORD_LIMIT];
__device__ uint8_t g_deviceFound[1];

__device__ __host__ bool next(uint8_t* length, uint8_t* word, uint32_t increment){
  uint32_t idx = 0;
  uint32_t add = 0;

  while(increment > 0 && idx < CONST_WORD_LIMIT){
    if(idx >= *length && increment > 0){
      increment--;
    }

    add = increment + word[idx];
    word[idx] = add % CONST_CHARSET_LENGTH;
    increment = add / CONST_CHARSET_LENGTH;
    idx++;
  }

  if(idx > *length){
    *length = idx;
  }

  if(idx > CONST_WORD_LENGTH_MAX){
    return false;
  }

  return true;
}

__global__ void md5Crack(uint8_t wordLength, uint8_t* charsetWord, uint32_t hash01, uint32_t hash02, uint32_t hash03, uint32_t hash04){
  uint32_t idx = (blockIdx.x * blockDim.x + threadIdx.x) * HASHES_PER_KERNEL;

  /* Thread variables */
  uint8_t threadCharsetWord[CONST_WORD_LIMIT];
  uint8_t threadWordLength;
  uint32_t threadHash01, threadHash02, threadHash03, threadHash04;

  /* Copy everything to local memory */
  memcpy(threadCharsetWord, charsetWord, CONST_WORD_LIMIT);
  memcpy(&threadWordLength, &wordLength, sizeof(uint8_t));

  /* Increment current word by thread index */
  next(&threadWordLength, threadCharsetWord, idx);

  for(uint32_t hash = 0; hash < HASHES_PER_KERNEL; hash++){

    md5Hash((uint8_t*)threadCharsetWord, threadWordLength, &threadHash01, &threadHash02, &threadHash03, &threadHash04);

    if(threadHash01 == hash01 && threadHash02 == hash02 && threadHash03 == hash03 && threadHash04 == hash04){
      memcpy(g_deviceCracked, threadCharsetWord, threadWordLength);
      g_deviceFound[0] = 1;
      break;
    }

    if(!next(&threadWordLength, threadCharsetWord, 1)){
      break;
    }
  }
}

int main(int argc, char* argv[]){
  /* Check arguments */
  if(argc != 2 || strlen(argv[1]) != 32){
    std::cout << argv[0] << " <md5_hash>" << std::endl;
    return -1;
  }

  /* Amount of available devices */
  int devices;
  ERROR_CHECK(hipGetDeviceCount(&devices));

  /* Sync type */
  ERROR_CHECK(hipSetDeviceFlags(hipDeviceScheduleSpin));

  /* Display amount of devices */
  std::cout << "Notice: " << devices << " device(s) found" << std::endl;

  /* Hash stored as u32 integers */
  uint32_t md5Hash[4];

  /* Parse argument */
  for(uint8_t i = 0; i < 4; i++){
    char tmp[16];

    strncpy(tmp, argv[1] + i * 8, 8);
    sscanf(tmp, "%x", &md5Hash[i]);
    md5Hash[i] = (md5Hash[i] & 0xFF000000) >> 24 | (md5Hash[i] & 0x00FF0000) >> 8 | (md5Hash[i] & 0x0000FF00) << 8 | (md5Hash[i] & 0x000000FF) << 24;
  }

  /* Fill memory */
  memset(g_word, 0, CONST_WORD_LIMIT);
  memset(g_cracked, 0, CONST_WORD_LIMIT);

  /* Current word length = minimum word length */
  g_wordLength = CONST_WORD_LENGTH_MIN;

  /* Main device */
  hipSetDevice(0);

  /* Time */
  hipEvent_t clockBegin;
  hipEvent_t clockLast;
  hipEvent_t clockProgressBegin;
  hipEvent_t clockProgressLast;

  hipEventCreate(&clockBegin);
  hipEventCreate(&clockLast);
  hipEventCreate(&clockProgressBegin);
  hipEventCreate(&clockProgressLast);
  hipEventRecord(clockBegin, 0);

  /* Current word is different on each device */
  uint8_t** words = new uint8_t*[devices];

  uint8_t founds[1] = {0};
  uint64_t counter = 0;

  for(int device = 0; device < devices; device++){
    hipSetDevice(device);

    /* Copy to each device */
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCracked), g_cracked, sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceFound), founds, sizeof(uint8_t), 0, hipMemcpyHostToDevice));

    /* Allocate on each device */
    ERROR_CHECK(hipMalloc((void**)&words[device], sizeof(uint8_t) * CONST_WORD_LIMIT));
  }

  while(true){
    bool result = false;

    hipEventRecord(clockProgressBegin, 0);
    for(int device = 0; device < devices; device++){
      hipSetDevice(device);

      /* Copy current data */
      ERROR_CHECK(hipMemcpy(words[device], g_word, sizeof(uint8_t) * CONST_WORD_LIMIT, hipMemcpyHostToDevice)); 

      /* Start kernel */
      md5Crack<<<TOTAL_BLOCKS, TOTAL_THREADS>>>(g_wordLength, words[device], md5Hash[0], md5Hash[1], md5Hash[2], md5Hash[3]);

      /* Global increment */
      result = next(&g_wordLength, g_word, TOTAL_THREADS * HASHES_PER_KERNEL * TOTAL_BLOCKS);
    }

    /* Display progress */
    float ms = 0;
    counter += TOTAL_THREADS * HASHES_PER_KERNEL * TOTAL_BLOCKS;
    hipEventRecord(clockProgressLast, 0);
    hipEventSynchronize(clockProgressLast);
    hipEventElapsedTime(&ms, clockProgressBegin, clockProgressLast);

    printf("\rNotice: currently counter %lu, time: %f ms, speed: %u hash/ms",
	   counter, ms, (unsigned int)((TOTAL_THREADS * HASHES_PER_KERNEL * TOTAL_BLOCKS) / ms));
    fflush(NULL);

    for(int device = 0; device < devices; device++){
      hipSetDevice(device);

      /* Synchronize now */
      hipDeviceSynchronize();

      /* Copy result */
      ERROR_CHECK(hipMemcpyFromSymbol(g_cracked, HIP_SYMBOL(g_deviceCracked), sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyDeviceToHost)); 
      ERROR_CHECK(hipMemcpyFromSymbol(founds, HIP_SYMBOL(g_deviceFound), sizeof(uint8_t), 0, hipMemcpyDeviceToHost));

      /* Check result */
      if(founds[0] == 1){
	std::cout << std::endl;
        std::cout << "Notice: cracked " << g_cracked << std::endl;
        break;
      }
    }

    if(!result || founds[0]){
      if(!result && !founds[0]){
	std::cout << std::endl;
        std::cout << "Notice: found nothing (host)" << std::endl;
      }

      break;
    }
  }

  for(int device = 0; device < devices; device++){
    hipSetDevice(device);

    /* Free on each device */
    hipFree((void**)words[device]);
  }

  /* Free array */
  delete[] words;

  /* Main device */
  hipSetDevice(0);

  float milliseconds = 0;

  hipEventRecord(clockLast, 0);
  hipEventSynchronize(clockLast);
  hipEventElapsedTime(&milliseconds, clockBegin, clockLast);

  std::cout << "Notice: computation time " << milliseconds << " ms" << std::endl;

  hipEventDestroy(clockBegin);
  hipEventDestroy(clockLast);
  hipEventDestroy(clockProgressBegin);
  hipEventDestroy(clockProgressLast);
}
